
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }


__global__ void foo(int *a, int N) {
 int i=blockIdx.x*blockDim.x+threadIdx.x;
    a[i]=i;
}

int main()
{
  int N=4097;
  int threads=128;
  int blocks=(N+threads-1)/threads;
  int *a;

  hipMallocManaged(&a,N*sizeof(int));
  foo<<<blocks,threads>>>(a, N);
  hipDeviceSynchronize();

  for(int i=0;i<10;i++)
    printf("%d\n",a[i]);

  cudaCheckError();
  return 0;
}

