#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <math_functions.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <glm/common.hpp>
#include <glm/glm.hpp>

#include <stdbool.h>

#include "driver/cuda_helper.h"
#include "utils.h"
#include "scene_data.h"

//#define EPSILON 0.0000001;

surface<void, cudaSurfaceType2D> surf;

union rgba_24
{
	uint1 b32;

	struct
	{
		unsigned  r : 8;
		unsigned  g : 8;
		unsigned  b : 8;
		unsigned  a : 8;
	};
};

__device__ inline bool
intersectTriangle(const glm::vec3 *vert, const scene::Ray &ray, glm::vec3& n, float& t)
{
	glm::vec3 v0v1 = vert[1] - vert[0];
	glm::vec3 v0v2 = vert[2] - vert[0];
	n = glm::normalize(glm::cross(v0v1, v0v2));
	glm::vec3 p_vec = glm::cross(ray.dir, v0v2);
	float det = glm::dot(v0v1, p_vec);
	if (det < 0.0000001)
		return false;

	float inv_det = __fdividef(1.f, det);
	glm::vec3 t_vec = ray.origin - vert[0];
	float u = glm::dot(t_vec, p_vec) * inv_det;
	if (u < 0 || u > 1) return false;

	glm::vec3 qvec = glm::cross(t_vec, v0v1);
	float v = glm::dot(ray.dir, qvec) * inv_det;
	if (v < 0 || u + v > 1) return false;

	t = glm::dot(v0v2, qvec) * inv_det;
	return true;
}

__device__ bool intersectSphere(const scene::Ray &r, float rad, glm::vec3 pos, float& t) {

	glm::vec3 op = pos - r.origin;
	float epsilon = 0.01f;
	float b = dot(op, r.dir);
	float disc = b*b - dot(op, op) + rad*rad;
	if (disc < 0)
		return 0;
	else
		//disc = sqrtf(disc);
		disc = __fsqrt_rn(disc);
	(t = b - disc) > epsilon ? t : ((t = b + disc) > epsilon ? t : 0);

	return t != 0;
}

__device__ inline bool
intersect(const scene::Ray& r,
	const struct scene::SceneData *const scene, glm::vec3& n, float& t, bool& light_emitter)
{
	glm::vec3 vertex[3];
	for (size_t m = 0; m < scene->meshes.size; ++m)
	{
		const scene::Mesh &mesh = scene->meshes.data[m];
		for (size_t i = 0; i < mesh.indices.size; i += 3)
		{
			for (size_t v = 0; v < 3; ++v)
			{
				tinyobj::index_t idx = mesh.indices.data[i + v];
				vertex[v].x = scene->vertices.data[3 * idx.vertex_index];
				vertex[v].y = scene->vertices.data[3 * idx.vertex_index + 1];
				vertex[v].z = scene->vertices.data[3 * idx.vertex_index + 2];
			}
			if (intersectTriangle(vertex, r, n, t))
				return true;

			if (intersectSphere(r, 0.5f, glm::vec3(0.0f, 0.2f, 0.0f), t))
			{
				light_emitter = true;
				return true;
			}
		}
	}
	return false;
}

#define M_PI 3.14159265359f

__device__ inline glm::vec3 radiance(scene::Ray& r,
	const struct scene::SceneData *const scene, hiprandState* rand_state, int is_static, int static_samples)
{
	glm::vec3 mask = glm::vec3(1.0f, 1.0f, 1.0f);
	glm::vec3 acc = glm::vec3(0.0f, 0.0f, 0.0f);

	const int max_bounces = 1 + is_static * (static_samples + 1);
	for (int b = 0; b < max_bounces; b++)
	{
		glm::vec3 normal;
		glm::vec3 oriented_normal;
		glm::vec3 color = glm::vec3(0.2f, 0.2f, 0.1f);
		// Light energy emission
		glm::vec3 emission = glm::vec3(2.0f);
		// For energy compensation on Russian roulette
		glm::vec3 thoughput = glm::vec3(1.0f);
		glm::vec3 mat_reflectance = glm::vec3(1.0f);
		float t = 100000;
		bool light_emitter = false;

		//float intersection = (float)intersect(r, scene, normal, t, light_emitter);
		if (intersect(r, scene, normal, t, light_emitter))
		{
			float cos_theta = glm::dot(normal, r.dir);
			oriented_normal = cos_theta < 0 ? normal : normal * -1.0f;

			//acc += mask * emission * (float)light_emitter * intersection;
			acc += mask * emission * (float)light_emitter * thoughput;

			float r2 = sqrtf(hiprand_uniform(rand_state));
			float r1 = 2.0f * M_PI * hiprand_uniform(rand_state);

			// Russian roulette
			float p = fmaxf(thoughput.x, fmaxf(thoughput.y, thoughput.z));
			if (r2 > p)
				return acc;

			thoughput *= 1 / p;

			float r2_squared = sqrtf(r2);

			glm::vec3 u = glm::normalize(glm::cross(fabs(oriented_normal.x) > .1 ? glm::vec3(0.0f, 1.0f, 0.0f) : glm::vec3(1.0f, 0.0f, 0.0f), oriented_normal));
			glm::vec3 v = glm::cross(oriented_normal, u);

			glm::vec3 d = glm::normalize(u * cos(r1) * r2_squared + v * sin(r1) * r2_squared + oriented_normal * sqrtf(1 - r2));

			r.origin += r.dir * t;

			r.origin += oriented_normal * 0.03f;
			r.dir = d;

			//mask *= intersection * color + (1.0f - intersection) * 1.0f;
			glm::vec3 BRDF = 2.0f * mat_reflectance * cos_theta * color;
			mask *= BRDF;
		}
	}

	return acc;
}

__global__ void
kernel(const unsigned int width, const unsigned int height,
	const unsigned int half_w, const unsigned int half_h,
	const scene::SceneData *const scene, unsigned int hash_seed,
	glm::vec3 offset, glm::vec3 dir_offset, int frame_nb, glm::vec3 *temporal_framebuffer, bool moved)
{
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	const unsigned int tid = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	union rgba_24 rgbx;
	rgbx.a = 0;

	hiprandState rand_state;
	hiprand_init(hash_seed + tid, 0, 0, &rand_state);

	struct scene::Camera *cam = scene->cam;
	float screen_dist = half_w / __tanf(cam->fov_x * 0.5);

	glm::vec3 look_at = glm::normalize(cam->dir + dir_offset);

	glm::vec3 cx = glm::vec3(width * cam->fov_x / height, 0.0f, 0.0f);
	glm::vec3 cy = glm::normalize(glm::cross(cx, look_at)) * cam->fov_x;

	glm::vec3 rad = glm::vec3(0.0f);
	scene::Ray r;
	r.dir = cx*((.25f + x) / width - .5f) + cy*((.25f + y) / height - .5f) + look_at;
	r.dir = glm::normalize(r.dir);
	r.origin = r.dir * 40.f + cam->position + offset / 10.f;

	int is_static = !moved;
	int static_samples = 1;
	int samples = 2 + is_static * static_samples;
	for (int i = 0; i < samples; i++)
		rad += radiance(r, scene, &rand_state, is_static, static_samples);

	rad /= samples;

	rad = glm::clamp(rad, 0.0f, 1.0f);

	int i = (height - y - 1)*width + x;
	temporal_framebuffer[i] *= is_static;
	temporal_framebuffer[i] += rad;

	rad = temporal_framebuffer[i] / (float)frame_nb;

	rgbx.r = rad.x * 255;
	rgbx.g = rad.y * 255;
	rgbx.b = rad.z * 255;

	surf2Dwrite(rgbx.b32,
		surf,
		x * sizeof(rgbx),
		y,
		hipBoundaryModeZero);
}

inline unsigned int WangHash(unsigned int a)
{
	a = (a ^ 61) ^ (a >> 16);
	a = a + (a << 3);
	a = a ^ (a >> 4);
	a = a * 0x27d4eb2d;
	a = a ^ (a >> 15);
	return a;
}

hipError_t
raytrace(hipArray_const_t array, const scene::SceneData *const scene,
	const unsigned int width, const unsigned int height, hipStream_t stream,
	glm::vec3 offset, glm::vec3 dir_offset, glm::vec3 *temporal_framebuffer, bool moved)
{
	static unsigned int seed = 0;

	if (moved)
		seed = 0;

	seed++;

	cudaBindSurfaceToArray(surf, array);

	// Register occupancy : nb_threads = regs_per_block / 32
	// Shared memory occupancy : nb_threads = shared_mem / 32
	// Block size occupancy 

	// TODO: We should get into account GPU info, such as number of registers,
	// shared memory size, warp size, etc...
	dim3 threads_per_block(16, 16);
	dim3 nb_blocks(width / threads_per_block.x, height / threads_per_block.y);

	if (nb_blocks.x > 0 && nb_blocks.y > 0)
		kernel << <nb_blocks, threads_per_block, 0, stream >> > (width, height,
			width / 2, height / 2, scene, WangHash(seed), offset, dir_offset, seed, temporal_framebuffer, moved);

	return hipSuccess;
}