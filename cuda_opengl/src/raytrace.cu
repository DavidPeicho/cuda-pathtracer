#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <math_functions.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <glm/common.hpp>
#include <glm/glm.hpp>

#include <stdbool.h>

#include "driver/cuda_helper.h"
#include "utils.h"
#include "scene_data.h"

//#define EPSILON 0.0000001;

surface<void, cudaSurfaceType2D> surf;

union rgba_24
{
	uint1 b32;

	struct
	{
		unsigned  r : 8;
		unsigned  g : 8;
		unsigned  b : 8;
		unsigned  a : 8;
	};
};

__device__ inline bool
intersectTriangle(const glm::vec3 *vert, const scene::Ray &ray, glm::vec3& n, float& t)
{
	glm::vec3 v0v1 = vert[1] - vert[0];
	glm::vec3 v0v2 = vert[2] - vert[0];
	n = glm::normalize(glm::cross(v0v1, v0v2));
	glm::vec3 p_vec = glm::cross(ray.dir, v0v2);
	float det = glm::dot(v0v1, p_vec);
	if (det < 0.0000001)
		return false;

	float inv_det = __fdividef(1.f, det);
	glm::vec3 t_vec = ray.origin - vert[0];
	float u = glm::dot(t_vec, p_vec) * inv_det;
	if (u < 0 || u > 1) return false;

	glm::vec3 qvec = glm::cross(t_vec, v0v1);
	float v = glm::dot(ray.dir, qvec) * inv_det;
	if (v < 0 || u + v > 1) return false;

	t = glm::dot(v0v2, qvec) * inv_det;
	return true;
}

__device__ bool intersectSphere(const scene::Ray &r, float rad, glm::vec3 pos, float& t) {

	glm::vec3 op = pos - r.origin;
	float epsilon = 0.01f;
	float b = dot(op, r.dir);
	float disc = b*b - dot(op, op) + rad*rad;
	if (disc < 0) return 0; else disc = sqrtf(disc);
	(t = b - disc) > epsilon ? t : ((t = b + disc) > epsilon ? t : 0);

	return t != 0;
}

__device__ inline bool
intersect(const scene::Ray& r,
	const struct scene::SceneData *const scene, glm::vec3& n, float& t)
{
	glm::vec3 vertex[3];
	for (size_t m = 0; m < scene->meshes.size; ++m)
	{
		const scene::Mesh &mesh = scene->meshes.data[m];
		for (size_t i = 0; i < mesh.indices.size; i += 3)
		{
			for (size_t v = 0; v < 3; ++v)
			{
				tinyobj::index_t idx = mesh.indices.data[i + v];
				vertex[v].x = scene->vertices.data[3 * idx.vertex_index];
				vertex[v].y = scene->vertices.data[3 * idx.vertex_index + 1];
				vertex[v].z = scene->vertices.data[3 * idx.vertex_index + 2];
			}
			if (intersectTriangle(vertex, r, n, t))
				return true;

			//if (intersectSphere(r, )
		}
	}
	return false;
}

#define M_PI 3.14159265359f

__device__ inline glm::vec3 radiance(scene::Ray& r,
	const struct scene::SceneData *const scene, hiprandState* rand_state)
{
	glm::vec3 mask = glm::vec3(1.0f, 1.0f, 1.0f);
	glm::vec3 acc = glm::vec3(0.0f, 0.0f, 0.0f);

	const int max_bounces = 2;
	for (int b = 0; b < max_bounces; b++)
	{
		glm::vec3 normal;
		glm::vec3 oriented_normal;
		glm::vec3 new_dir;
		glm::vec3 inter_point;
		glm::vec3 color = glm::vec3(0.2f, 0.2f, 0.1f);
		glm::vec3 emission = glm::vec3(1.0f);
		float t = 100000;

		if (intersect(r, scene, normal, t))
		{
			oriented_normal = glm::dot(normal, r.dir) < 0 ? normal : normal * -1.0f;

			acc += mask * emission;

			float r1 = 2.0f * M_PI * hiprand_uniform(rand_state);
			float r2 = hiprand_uniform(rand_state);

			float r2_squared = sqrtf(r2);

			glm::vec3 u = glm::normalize(glm::cross(fabs(oriented_normal.x) > .1 ? glm::vec3(0.0f, 1.0f, 0.0f) : glm::vec3(1.0f, 0.0f, 0.0f), oriented_normal));
			glm::vec3 v = glm::cross(oriented_normal, u);

			glm::vec3 d = glm::normalize(u * cos(r1) * r2_squared + v * sin(r1) * r2_squared + oriented_normal * sqrtf(1 - r2));

			r.origin += r.dir * t;

			r.origin += oriented_normal * 0.03f;
			r.dir = d;

			mask *= color;
		}
	}

	return acc;
}

__global__ void
kernel(const unsigned int width, const unsigned int height,
	const unsigned int half_w, const unsigned int half_h,
	const scene::SceneData *const scene, unsigned int hash_seed,
	glm::vec3 offset, glm::vec3 dir_offset, int frame_nb, glm::vec3 *temporal_framebuffer)
{
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	const unsigned int tid = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	union rgba_24 rgbx;
	rgbx.a = 0;

	hiprandState rand_state;
	hiprand_init(hash_seed + tid, 0, 0, &rand_state);

	struct scene::Camera *cam = scene->cam;
	float screen_dist = half_w / __tanf(cam->fov_x * 0.5);

	glm::vec3 cx = glm::vec3(width * cam->fov_x / height, 0.0f, 0.0f);
	glm::vec3 cy = glm::normalize(glm::cross(cx, cam->dir)) * cam->fov_x;

	glm::vec3 rad = glm::vec3(0.0f);
	scene::Ray r;
	r.dir = cx*((.25f + x) / width - .5f) + cy*((.25f + y) / height - .5f) + glm::normalize(cam->dir + dir_offset);
	r.dir = glm::normalize(r.dir);
	r.origin = r.dir * 40.f + cam->position - offset / 10.f;

	int samples = 2;
	for (int i = 0; i < samples; i++)
		rad += radiance(r, scene, &rand_state);

	rad /= samples;

	rad = glm::clamp(rad, 0.0f, 1.0f);

	int i = (height - y - 1)*width + x;
	//temporal_framebuffer[i] *= (frame_nb - 1);
	temporal_framebuffer[i] += rad;

	rad = temporal_framebuffer[i] / (float)frame_nb;

	rgbx.r = rad.x * 255;
	rgbx.g = rad.y * 255;
	rgbx.b = rad.z * 255;

	surf2Dwrite(rgbx.b32,
		surf,
		x * sizeof(rgbx),
		y,
		hipBoundaryModeZero);
}

inline unsigned int WangHash(unsigned int a)
{
	a = (a ^ 61) ^ (a >> 16);
	a = a + (a << 3);
	a = a ^ (a >> 4);
	a = a * 0x27d4eb2d;
	a = a ^ (a >> 15);
	return a;
}

hipError_t
raytrace(hipArray_const_t array, const scene::SceneData *const scene,
	const unsigned int width, const unsigned int height, hipStream_t stream, glm::vec3 offset, glm::vec3 dir_offset, glm::vec3 *temporal_framebuffer)
{
	static unsigned int seed = 0;
	seed++;

	cudaBindSurfaceToArray(surf, array);

	// Register occupancy : nb_threads = regs_per_block / 32
	// Shared memory occupancy : nb_threads = shared_mem / 32
	// Block size occupancy 

	// TODO: We should get into account GPU info, such as number of registers,
	// shared memory size, warp size, etc...
	dim3 threads_per_block(16, 16);
	dim3 nb_blocks(width / threads_per_block.x, height / threads_per_block.y);

	if (nb_blocks.x > 0 && nb_blocks.y > 0)
		kernel << <nb_blocks, threads_per_block, 0, stream >> > (width, height,
			width / 2, height / 2, scene, WangHash(seed), offset, dir_offset, seed, temporal_framebuffer);

	return hipSuccess;
}